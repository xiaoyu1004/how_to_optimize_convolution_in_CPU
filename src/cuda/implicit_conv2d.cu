#include "hip/hip_runtime.h"
#include "implicit_conv2d.h"

template <typename Tin, typename Tw, typename Tacc, typename Tout>
__global__ void implicit_conv2d_kernel_v1(int input_n, int input_c, int input_h, int input_w,
                                          int output_c, int kernel_h, int kernel_w,
                                          int output_h, int output_w,
                                          int stride_h, int stride_w,
                                          int pad_h, int pad_w,
                                          int dilation_h, int dilation_w,
                                          int group_count,
                                          const Tin *x, const Tw *w, const Tacc *bias, Tout *y)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= input_n * output_h * output_w || iy >= output_c)
        return;

    int oc = iy;
    int n = ix / (output_h * output_w);
    int oh = ix % (output_h * output_w) / output_w;
    int ow = ix % (output_h * output_w) % output_w;

    Tacc result = static_cast<Tacc>(0);
    int k = input_c * kernel_h * kernel_w;
    for (int i = 0; i < k; ++i)
    {
        // calc index
        int ic = i / (kernel_h * kernel_w);
        int kh = i % (kernel_h * kernel_w) / kernel_w;
        int kw = i % (kernel_h * kernel_w) % kernel_w;

        int ih = oh * stride_h + kh - pad_h;
        int iw = ow * stride_w + kw - pad_w;
        if (ih < 0 || ih >= input_h || iw < 0 || iw >= input_w)
            continue;

        int x_idx = ((n * input_c + ic) * input_h + ih) * input_w + iw;
        int w_idx = ((oc * input_c + ic) * kernel_h + kh) * kernel_w + kw;
        result += x[x_idx] * w[w_idx];
    }
    if (bias)
    {
        result += bias[oc];
    }
    int y_idx = ((n * output_c + oc) * output_h + oh) * output_w + ow;
    y[y_idx] = result;
}

template <const int BLOCK_SIZE_M, const int BLOCK_SIZE_N, const int BLOCK_SIZE_K,
          typename Tin, typename Tw, typename Tacc, typename Tout>
__global__ void implicit_conv2d_kernel_v2(int input_n, int input_c, int input_h, int input_w,
                                          int output_c, int kernel_h, int kernel_w,
                                          int output_h, int output_w,
                                          int stride_h, int stride_w,
                                          int pad_h, int pad_w,
                                          int dilation_h, int dilation_w,
                                          int group_count,
                                          const Tin *x, const Tw *w, const Tacc *bias, Tout *y)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int m = output_c;
    int n = output_h * output_w;
    int k = input_c * kernel_h * kernel_w;

    __shared__ Tw SLB_W[BLOCK_SIZE_M][BLOCK_SIZE_K];
    __shared__ Tin SLB_X[BLOCK_SIZE_K][BLOCK_SIZE_N];

    for (int idx = 0; idx < k; idx += BLOCK_SIZE_K)
    {
        // load kernel from gmem to smem
        Tw val_w = static_cast<Tw>(0);
        if ((by * BLOCK_SIZE_M + ty) < m && (idx + tx) < k)
        {
            int n = by * BLOCK_SIZE_M + ty;
            int ic = (idx + tx) / (kernel_h * kernel_w);
            int rest_offset = (idx + tx) % (kernel_h * kernel_w);
            val_w = w[(n * input_c + ic) * kernel_h * kernel_w + rest_offset];
        }

        // load input from gmem to smem
        if ((idx + ty) < k && ())
    }
}

template <typename Tin, typename Tw, typename Tacc, typename Tout>
void implicit_conv2d_gpu(int input_n, int input_c, int input_h, int input_w,
                         int output_c, int kernel_h, int kernel_w,
                         int stride_h, int stride_w,
                         int pad_h, int pad_w,
                         int dilation_h, int dilation_w,
                         int group_count,
                         const Tin *x, const Tw *w, const Tacc *bias, Tout *y)
{
    int khd = (kernel_h - 1) * dilation_h + 1;
    int kwd = (kernel_w - 1) * dilation_w + 1;
    int output_h = (input_h - khd + 2 * pad_h) / stride_h + 1;
    int output_w = (input_w - kwd + 2 * pad_w) / stride_w + 1;

    int m = output_c;
    int n = input_n * output_h * output_w;

    constexpr int BLOCK_SIZE_M = 32;
    constexpr int BLOCK_SIZE_N = 32;
    constexpr int BLOCK_SIZE_K = 32;

    dim3 dimBlock(BLOCK_SIZE_N, BLOCK_SIZE_M);
    dim3 dimGrid((n + BLOCK_SIZE_N - 1) / BLOCK_SIZE_N, (m + BLOCK_SIZE_M - 1) / BLOCK_SIZE_M);
    implicit_conv2d_kernel<Tin, Tw, Tacc, Tout><<<dimGrid, dimBlock>>>(input_n, input_c, input_h, input_w,
                                                                       output_c, kernel_h, kernel_w,
                                                                       output_h, output_w,
                                                                       stride_h, stride_w,
                                                                       pad_h, pad_w,
                                                                       dilation_h, dilation_w,
                                                                       group_count,
                                                                       x, w, bias, y);
}

template void implicit_conv2d_gpu<float, float, float, float>(int input_n, int input_c, int input_h, int input_w,
                                                              int output_c, int kernel_h, int kernel_w,
                                                              int stride_h, int stride_w,
                                                              int pad_h, int pad_w,
                                                              int dilation_h, int dilation_w,
                                                              int group_count,
                                                              const float *x, const float *w, const float *bias, float *y);