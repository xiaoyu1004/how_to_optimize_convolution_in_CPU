
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void bcast(int arg) 
{
    int laneId = threadIdx.x & 0x1f;
    int value;
    if (laneId == 0)        // Note unused variable for
        value = arg;        // all threads except lane 0
    value = __shfl_sync(0xffffffff, value, 0);   // Synchronize all threads in warp, and get "value" from lane 0
    if (value != arg)
        printf("Thread %d failed.\n", threadIdx.x);
    else
        printf("Thread %d get val success val: %d.\n", threadIdx.x, value);
}

// int main() 
// {
//     bcast<<< 1, 32 >>>(1234);
//     cudaDeviceSynchronize();

//     return 0;
// }

__global__ void scan4() 
{
    int laneId = threadIdx.x & 0x1f;
    // Seed sample starting value (inverse of lane ID)
    int value = 31 - laneId;

    // Loop to accumulate scan within my partition.
    // Scan requires log2(n) == 3 steps for 8 threads
    // It works by an accumulated sum up the warp
    // by 1, 2, 4, 8 etc. steps.
    for (int i=1; i<=4; i*=2) {
        // We do the __shfl_sync unconditionally so that we
        // can read even from threads which won't do a
        // sum, and then conditionally assign the result.
        int n = __shfl_up_sync(0xffffffff, value, i, 8);
        if ((laneId & 7) >= i)
            value += n;
    }

    printf("Thread %d final value = %d\n", threadIdx.x, value);
}

// int main() 
// {
//     scan4<<< 1, 32 >>>();
//     cudaDeviceSynchronize();

//     return 0;
// }


__global__ void warpReduce() 
{
    int laneId = threadIdx.x & 0x1f;
    // Seed starting value as inverse lane ID
    int value = 31 - laneId;

    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    printf("Thread %d final value = %d\n", threadIdx.x, value);
}

int main() 
{
    warpReduce<<< 1, 32 >>>();
    hipDeviceSynchronize();

    return 0;
}

